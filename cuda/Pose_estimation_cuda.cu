#include "hip/hip_runtime.h"
#include "Pose_estimation_cuda.h"

namespace kinectfusion{
    
__device__
Vector3f Vector4fToVector3f(Vector4f vertex){
    Vector3f output;
    output.x() = vertex.x();
    output.y() = vertex.y();
    output.z() = vertex.z();
    return output;
}
__device__
Vector4f Vector3fToVector4f(Vector3f vertex){
    Vector4f output;
    output.x() = vertex.x();
    output.y() = vertex.y();
    output.z() = vertex.z();
    output.w() = 1.0;
    return output;
}

__device__
Vector3f TransformToVertex(Vector3f vertex, Eigen::Matrix4f Transformation)
{
    return Transformation.block(0, 0, 3, 3) * vertex + Transformation.block(0,3,3,1);

}

__device__
Vector3f TransformToNormal(Vector3f normal, Eigen::Matrix4f Transformation)
{
    return Transformation.block(0, 0, 3, 3) * normal;

}

__global__ 
void data_association_kernel(           const Vertex* frame_data,
                                        const Matrix3f* Intrinsics,
                                        const unsigned int width,
                                        const unsigned int height,
                                        Match* matches,
                                        int* match_count,
                                        unsigned int frame_data_size,
                                        const Matrix4f previous_pose,
                                        const Matrix4f current_pose)
{
    float fX = Intrinsics[0](0,0);
    float fY = Intrinsics[0](1,1);
    float cX = Intrinsics[0](0,2);
    float cY = Intrinsics[0](1,2);
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // int idx = y * width + x;
    if (idx >= frame_data_size) {
        return;
    }

    Vector3f current_vertex = Vector4fToVector3f(frame_data[idx].position);
    Vector3f current_normal = frame_data[idx].normal;

    // avoid redundant caculation to speed up 
    if (!isnan(current_normal[0]) && !isnan(current_normal[1]) && !isnan(current_normal[2]) &&
        !isnan(current_vertex[0]) && !isnan(current_vertex[0]) && !isnan(current_vertex[0]) &&
        current_normal[0] != MINF && current_normal[2] != MINF && current_normal[2] != MINF &&
        current_vertex[0] != MINF && current_vertex[2] != MINF && current_vertex[2] != MINF)
    {
        // transfrom to camera coordinate v_k-1
        Eigen::MatrixXf tmpTransformation = previous_pose.inverse() * current_pose;
        Eigen::Vector3f current_camera_vertex = TransformToVertex(current_vertex, tmpTransformation);            
            // back-project to pixel v_k-1
        Eigen::Vector2i point;
        point.x() = std::round(current_camera_vertex.x() * fX / current_camera_vertex.z() + cX);
        point.y() = std::round(current_camera_vertex.y() * fY / current_camera_vertex.z() + cY);

        // check if pixel still in image
        // && current_camera_vertex.z() >= 0
        if(point.x() >= 0 && point.y() >= 0 && point.x() < width && point.y() < height && current_camera_vertex.z() >= 0){
            //cacluate v
            int previous_idx = point.y() * width + point.x();
            // i means point in frame k
            // previous means point in frame k-1
            int match_index = atomicAdd(match_count, 1);
            matches[match_index].cur_idx = idx;
            matches[match_index].prv_idx = previous_idx;            

        }                              
    }
}
    
void data_association_cuda(     const std::vector<Vertex>& frame_data,
                                const Matrix3f& Intrinsics,
                                const unsigned int& width,
                                const unsigned int& height,
                                std::unordered_map<int, int>& matches,
                                const Eigen::MatrixXf& previous_pose,
                                const Eigen::MatrixXf& current_pose
                                )
{
    // Allocate memory on the GPU + Copy data from host to device
    Vertex* d_frame_data;
    hipMalloc(&d_frame_data, sizeof(Vertex) * frame_data.size());
    hipMemcpy(d_frame_data, frame_data.data(), sizeof(Vertex) * frame_data.size(), hipMemcpyHostToDevice);
    
    Matrix3f* d_Intrinsics;
    hipMalloc(&d_Intrinsics, sizeof(Matrix3f));
    hipMemcpy(d_Intrinsics, &Intrinsics, sizeof(Matrix3f), hipMemcpyHostToDevice);

    Match* d_matches;
    int* d_match_count;
    hipMalloc(&d_matches, frame_data.size() * sizeof(Match));
    hipMalloc(&d_match_count, sizeof(int));
    int match_count = 0;
    hipMemcpy(d_match_count, &match_count, sizeof(int), hipMemcpyHostToDevice);    
    // cout << "d_frame_data is " << endl << d_frame_data-> << endl;
    cout << "frame_data is " << endl << frame_data[120840].position << endl;
    
    
    // Launch the kernel
    // dim3 threads(32, 32);
    // dim3 blocks(std::ceil(width / threads.x), std::ceil(height / threads.y));
    // dim3 threads(256);
    // dim3 blocks((frame_data.size() + threads.x - 1) / threads.x);    
    dim3 grid(ceil(frame_data.size() / (float)32), 1, 1);
    dim3 block(32, 1, 1);
    data_association_kernel <<<grid, block>>> (d_frame_data, d_Intrinsics, width, height, d_matches, d_match_count, frame_data.size(), previous_pose, current_pose);
    hipDeviceSynchronize();
    // Copy data from device to host

    hipMemcpy(&match_count, d_match_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(matches.data(), d_matches, match_count * sizeof(Match), hipMemcpyDeviceToHost);

    Match* h_matches = new Match[h_match_count];
    hipMemcpy(h_matches, d_matches, sizeof(Match) * matches.size(), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    cout << "matches check: " << h_matches << endl;
    cout << "check match_count_gpu: " << d_match_count << endl;
    cout << "match count is: " << h_match_count<< endl;

    // matches.clear();
    // for (int i = 0; i < *d_match_count; i++) {
    // matches[h_matches[i].cur_idx] = h_matches[i].prv_idx;
    // }
    
    // Free the GPU memory

    hipFree(d_frame_data);
    hipFree(d_matches);
    hipFree(d_match_count);
    delete[] h_matches;
}

}